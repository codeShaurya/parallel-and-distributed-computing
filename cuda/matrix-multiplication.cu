#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

_global_ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

int main(int argc, char const *argv[])
{
    int m, n, k;
    printf("please type in m n and k\n");
    scanf("%d %d %d", &m, &n, &k);

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **)&h_a, sizeof(int) * m * n);
    hipHostMalloc((void **)&h_b, sizeof(int) * n * k);
    hipHostMalloc((void **)&h_c, sizeof(int) * m * k);
    hipHostMalloc((void **)&h_cc, sizeof(int) * m * k);

    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, sizeof(int) * m * n);
    hipMalloc((void **)&d_b, sizeof(int) * n * k);
    hipMalloc((void **)&d_c, sizeof(int) * m * k);

    hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
    hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            printf("%d ",h_c[i*m+j]);
        }
        printf("\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}

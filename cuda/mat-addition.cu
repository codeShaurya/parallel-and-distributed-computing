#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void gpu_matrixadd(int *a, int *b, int *c, int N)
{

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int index = row * N + col;

  if (col < N && row < N)
    c[index] = a[index] + b[index];
}

void cpu_matrixadd(int *a, int *b, int *c, int N)
{

  int index;
  for (int col = 0; col < N; col++)
    for (int row = 0; row < N; row++)
    {
      index = row * N + col;
      c[index] = a[index] + b[index];
    }
}

int main(int argc, char *argv[])
{

  char key;

  int i, j; // loop counters

  int Grid_Dim_x = 1, Grid_Dim_y = 1;   //Grid structure values
  int Block_Dim_x = 1, Block_Dim_y = 1; //Block structure values

  int noThreads_x, noThreads_y; // number of threads available in device, each dimension
  int noThreads_block;          // number of threads in a block

  int N = 10; // size of array in each dimension
  int *a, *b, *c, *d;
  int *dev_a, *dev_b, *dev_c;
  int size; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms;   // which is applicable for asynchronous code also

  /* --------------------ENTER INPUT PARAMETERS AND DATA -----------------------*/

  do
  { // loop to repeat complete program

  __global__ void input_parameter(sizeof(x), sizeof(y), nub_block))


		x = (int*) malloc(size);		//this time use dynamically allocated memory for arrays on host
  y = (int *)malloc(size);
  nub_block = (int *)malloc(size); // results from GPU
                                   // results from CPU

  for (i = 0; i < N; i++) // load arrays with some numbers
    for (j = 0; j < N; j++)
    {
      a[i * N + j] = i;
      b[i * N + j] = i;
    }
  }
  }
